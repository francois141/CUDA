
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

template<typename T> __host__ __device__ inline void op(T &a,T &b)
{
  a = a + b;
}

__global__ void add(int n,float *x, float *y)
{
  int stride = blockDim.x * gridDim.x;
  int index  = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = index; i < n;i+=stride)
  {
    op<float>(y[i],x[i]);
  }
}

void addNormal(int n,float *x,float *y)
{
  for(int i = 0; i <n;i++) {
    op<float>(x[i],y[i]);
  }
}

__global__ void reverseFixedArray(int *d, int n)
{
  __shared__ int s[512];
  int t = threadIdx.x;
  int tr = n - t - 1;
  s[t] = d[t];
  __syncthreads(); // We need to put a barrier here
  d[t] = s[tr];
}

int main(void)
{
  /** CHECK DEVICES FIRST **/

  int numberDevices;
  hipGetDeviceCount(&numberDevices);

  for(int i = 0; i < numberDevices;i++)
  {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop,i);

    if(err != hipSuccess)
      std::cout << "There is an error ! " << std::endl;

    std::cout << "Device : " << i << std::endl;
    std::cout << "Name : " << prop.name <<  std::endl;
    std::cout << "Total global memory : " << prop.totalGlobalMem << std::endl;
    std::cout << "Max Threads per blocks : " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Major : " << prop.major << std::endl;
    std::cout << "Minor : " << prop.minor << std::endl;
    std::cout << std::endl;
  }

  /** TEST CUDA WITH A SIMPLE CODE **/

  int N = 1<<20;
  float *x, *y;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEventRecord(start);

  dim3 gridSize = dim3(1024,0,0);

  add<<<N/1024, gridSize>>>(N, x, y);
  
  hipEventRecord(stop);

  hipDeviceSynchronize();

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Time taken with GPU : " << milliseconds << " ms" << std::endl;

  hipFree(x);
  hipFree(y);

  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  auto start2 = std::chrono::steady_clock::now();
  addNormal(N,x,y);
  auto end2 = std::chrono::steady_clock::now();

  auto diff2 = end2 - start2;
  std::cout << "Time taken with CPU : " << std::chrono::duration <double, std::milli> (diff2).count() << " ms" << std::endl;

  constexpr unsigned int n = 512;
  int array[n];
  for(int i = 0; i < n;i++) {
    array[i] = i;
  }


  int *hipArray;
  hipMalloc(&hipArray,n*sizeof(int));

  hipMemcpy(hipArray,array,n*sizeof(int),hipMemcpyHostToDevice);
  reverseFixedArray<<<1,n>>>(hipArray,n);
  hipMemcpy(array,hipArray,n*sizeof(int),hipMemcpyDeviceToHost);

  std::cout << array[0] << " " << array[511] << std::endl;

  // Test the same but with a new stream

  int array2[n];
  for(int i = 0; i < n;i++) {
    array2[i] = i;
  }

  hipStream_t stream;
  hipStreamCreate(&stream);


  int *cudaArray2;
  hipMalloc(&cudaArray2,n*sizeof(int));

  hipMemcpyAsync(cudaArray2,array2,n*sizeof(int),hipMemcpyHostToDevice,stream);
  reverseFixedArray<<<1,n,0,stream>>>(cudaArray2,n);
  hipMemcpyAsync(array2,cudaArray2,n*sizeof(int),hipMemcpyDeviceToHost,stream);

  hipStreamDestroy(stream);

  std::cout << array2[0] << " " << array2[511] << std::endl;


  return 0;
}
