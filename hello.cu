
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

template<typename T> __host__ __device__ inline void op(T &a,T &b)
{
  a = a + b;
}

__global__ void add(int n,float *x, float *y)
{
  int stride = blockDim.x * gridDim.x;
  int index  = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = index; i < n;i+=stride)
  {
    op<float>(y[i],x[i]);
  }
}

void addNormal(int n,float *x,float *y)
{
  for(int i = 0; i <n;i++) {
    op<float>(x[i],y[i]);
  }
}

int main(void)
{
  /** CHECK DEVICES FIRST **/

  int numberDevices;
  hipGetDeviceCount(&numberDevices);

  for(int i = 0; i < numberDevices;i++)
  {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop,i);

    if(err != hipSuccess)
      std::cout << "There is an error ! " << std::endl;

    std::cout << "Device : " << i << std::endl;
    std::cout << "Name : " << prop.name <<  std::endl;
    std::cout << "Total global memory : " << prop.totalGlobalMem << std::endl;
    std::cout << "Max Threads per blocks : " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Major : " << prop.major << std::endl;
    std::cout << "Minor : " << prop.minor << std::endl;
    std::cout << std::endl;
  }

  /** TEST CUDA WITH A SIMPLE CODE **/

  int N = 1<<20;
  float *x, *y;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEventRecord(start);

  dim3 gridSize = dim3(1024,0,0);
  add<<<N/1024, gridSize>>>(N, x, y);
  
  hipEventRecord(stop);

  hipDeviceSynchronize();

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Time take with GPU : " << milliseconds << " ms" << std::endl;

  hipFree(x);
  hipFree(y);

  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  auto start2 = std::chrono::steady_clock::now();
  addNormal(N,x,y);
  auto end2 = std::chrono::steady_clock::now();

  auto diff2 = end2 - start2;
  std::cout << "Time take with CPU : " << std::chrono::duration <double, std::milli> (diff2).count() << " ms" << std::endl;

  return 0;
}
